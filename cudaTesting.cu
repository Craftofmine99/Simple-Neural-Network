#include "hip/hip_runtime.h"
#include "src/NeuralNetwork.hpp"
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

using namespace std;
/**
 * For testing the NeuralNetwork
 */

__device__
double sigmoidDevice(double x)
{
	return (1.0f / (1.0f + exp(-x)));
}

__global__
void forCuda(double *members, int *layerSizes, double* answers, double *input, int otherSize, int numInput, int memberSize)
{
    int threadIndex = threadIdx.x;

    double *previous = new double[numInput];
    double *result = new double[1];
    for(int i = 0 ; i < numInput ; i++)
        previous[i] = input[i];

    int index,tempIndex;

    for(int a = threadIndex*2 ; a < threadIndex*2+2 ; a++)
    {
        index = memberSize*a;
        for(int b = 0 ; b < otherSize ; b++)
        {
            delete [] result;
            result = new double[layerSizes[b]];
            for(int c = 0 ; c < layerSizes[b] ; c++)
            {
                double temp = 0.0;
                if(b == 0)
                {
                    tempIndex = index + c*numInput;
                    for(int d = 0 ; d < numInput ; d++)
                        temp += previous[d] * members[tempIndex + d];
                }
                else
                {
                    tempIndex = index + c*layerSizes[b-1];
                    for(int d = 0 ; d < layerSizes[b-1] ; d++)
                        temp += previous[d] * members[tempIndex + d];
                }
                result[c] = sigmoidDevice(temp);
            }
            if(b == 0)
                index += layerSizes[b]*numInput;
            else
                index += layerSizes[b]*layerSizes[b-1];
            delete [] previous;
            previous = new double[layerSizes[b]];
            for(int c = 0 ; c < layerSizes[b] ; c++)
                previous[c] = result[c];
        }
        for(int i = 0 ; i < layerSizes[otherSize-1] ; i++)
            answers[a + i] = result[i];
    }
    delete [] previous;
    delete [] result;
}

int main()
{
    vector<int> innerNodes = vector<int>();
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);
    innerNodes.emplace_back(64);

    int numInput = 768;
    int numOutput = 1;
    int numMembers = 512;

    NeuralNetwork myNetwork = NeuralNetwork(numInput, innerNodes, numOutput, true);
    myNetwork.init(numMembers);

    vector<double> input = vector<double>(numInput, 0.5f);

    hipProfilerStart();

    // Allocate memory
    double *members;
    int *lSizes;
    double *answers;
    hipMallocManaged(&answers, (numMembers*numOutput)*sizeof(double));

    int membersSize = 0;
    vector<vector<vector<double>>> tempMember;
    for(int a = 0 ; a < numMembers ; a++)
    {
        tempMember = myNetwork.getTableOf(a);
        for(int b = 0 ; b < tempMember.size() ; b++)
            for(int c = 0 ; c < tempMember[b].size() ; c++)
                membersSize += tempMember[b][c].size();
    }
    hipMallocManaged(&members, membersSize*sizeof(double));

    int size = 0;
    for(int a = 0 ; a < tempMember.size() ; a++)
        for(int b = 0 ; b < tempMember[a].size() ; b++)
            size += tempMember[a][b].size();

    vector<int> layerSizes = vector<int>(innerNodes);
    layerSizes.emplace_back(numOutput);
    hipMallocManaged(&lSizes,layerSizes.size()*sizeof(int));
    for(int i = 0 ; i < layerSizes.size() ; i++)
        lSizes[i] = layerSizes[i];

    cout << "Size of array \"members\" : " << membersSize << endl;

    vector<bool> toNext = vector<bool>();

    for (; toNext.size() < numMembers;)
    {
        toNext.push_back(false);
        toNext.push_back(true);
        toNext.push_back(false);
        toNext.push_back(false);
    }

    toNext.shrink_to_fit();

    cout << "Members : " << numMembers << endl;

    int avgMoves = 40;
    int numMovesChecked = 100;

    cout << "Moves per \"game\" : " << avgMoves*numMovesChecked << endl;
    for (int i = 0; i < 16; i++)
    {
        clock_t begin_time = clock();

        // Fill it
        int index = 0;
        for(int a = 0 ; a < numMembers ; a++)
        {
            tempMember = myNetwork.getTableOf(a);
            for(int b = 0 ; b < tempMember.size() ; b++)
                for(int c = 0 ; c < tempMember[b].size() ; c++)
                    for(int d = 0 ; d < tempMember[b][c].size() ; d++)
                        members[index++] = tempMember[b][c][d];
        }

        double *tempInput;
        hipMallocManaged(&tempInput, numInput*sizeof(double));

        for(int a = 0 ; a < avgMoves*numMovesChecked ; a++)
        {
            // hipFree(answers);
            // hipMallocManaged(&answers, (numMembers)*sizeof(double));

            for(int b = 0 ; b < numInput ; b++)
                tempInput[b] = myNetwork.dist(myNetwork.e2);

            // cout << "Start GPU" << endl;
            forCuda<<<1,numMembers/2>>>(members, lSizes, answers, tempInput, layerSizes.size(), numInput, size);
            hipDeviceSynchronize();
            // cout << "End GPU" << endl;

            // cout << "Answer sum : " << (a+1) << endl;
            // double temp = 0.0;
            // for(int b = 0 ; b < numMembers ; b++)
            //     temp += answers[b];
            // cout << temp << endl;
        }

        hipFree(tempInput);

        if (myNetwork.nextGen(toNext))
        {

            clock_t end_time = clock();
            int hours = floor((float(end_time - begin_time) / CLOCKS_PER_SEC) / 3600.0f);
            int minuets = floor((float(end_time - begin_time) / CLOCKS_PER_SEC - hours * 60) / 60.0f);
            int seconds = float(end_time - begin_time) / CLOCKS_PER_SEC - hours * 60.0f - minuets * 60.0f;
            cout << (i + 1) << "\t" << hours << " : " << minuets << " : " << seconds << endl;
            cout << "---------------" << endl;
        }
        else
            break;
    }


    hipFree(members);
    hipFree(lSizes);
    hipFree(answers);

    hipProfilerStop();

    return 0;
}